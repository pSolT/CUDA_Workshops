#include "hip/hip_runtime.h"
/*
 * Image.cpp
 *
 *  Created on: 6 gru 2015
 *      Author: pSolT
 */

#include "Image.cuh"

Image::Image()
{

}

Image::~Image()
{
	free(h_greyImage__);
	free(h_rgbaImage__);
	hipFree(d_rgbaImage__);
	hipFree(d_greyImage__);
}

Image& Image::ApplyRGBAFilter(RGBAFilter * filter)
{
	int numPixels = GetColumnsCount() * GetRowsCount();
	uchar4 * result;
	hipMalloc(&result, sizeof(uchar4) * numPixels);
	hipMemcpy(d_rgbaImage__, h_rgbaImage__, sizeof(uchar4)*numPixels, hipMemcpyHostToDevice);
	filter->Apply(d_rgbaImage__, result, GetRowsCount(), GetColumnsCount());
	hipMemcpy(h_rgbaImage__, result, sizeof(uchar4)*numPixels, hipMemcpyDeviceToHost);
	return *this;
}

Image& Image::ApplyGreyscaleFilter(GreyscaleFilter * filter)
{
	int numPixels = GetColumnsCount() * GetRowsCount();
	unsigned char * result;
	hipMalloc(&result, sizeof(unsigned char) * numPixels);
	hipMemcpy(d_greyImage__, h_greyImage__, sizeof(unsigned char)*numPixels, hipMemcpyHostToDevice);
	filter->Apply(d_greyImage__, result, GetRowsCount(), GetColumnsCount());
	hipMemcpy(h_greyImage__, result, sizeof(unsigned char)*numPixels, hipMemcpyDeviceToHost);
	return *this;
}

Image* Image::Load(const std::string &filename)
{
	Image * result = new Image();
	cv::Mat image;
	image = cv::imread(filename.c_str(), CV_LOAD_IMAGE_COLOR);
	if (image.empty()) {
		std::cerr << "Couldn't open file: " << filename << std::endl;
		exit(1);
	}

	// Convert image from default OpenCV color space  to RGBA
	cv::cvtColor(image, result->imageRGBA, CV_BGR2RGBA);

	//allocate memory for the output
	result->imageGrey.create(image.rows, image.cols, CV_8UC1);

	//This shouldn't ever happen given the way the images are created
	//at least based upon my limited understanding of OpenCV, but better to check
	if (!result->imageRGBA.isContinuous() || !result->imageGrey.isContinuous()) {
		std::cerr << "Images aren't continuous!! Exiting." << std::endl;
		exit(1);
	}


	result->h_rgbaImage__ = (uchar4 *) result->imageRGBA.ptr<unsigned char>(0);
	result->h_greyImage__ = result->imageGrey.ptr<unsigned char>(0);

	const size_t numPixels = result->GetRowsCount() * result->GetColumnsCount();
	//allocate memory on the device for both input and output
	checkCudaErrors(hipMalloc(&result->d_rgbaImage__, sizeof(uchar4) * numPixels));
	checkCudaErrors(hipMalloc(&result->d_greyImage__, sizeof(unsigned char) * numPixels));

	//copy input array to the GPU
	checkCudaErrors(
			hipMemcpy(result->d_rgbaImage__, result->h_rgbaImage__, sizeof(uchar4) * numPixels,
					hipMemcpyHostToDevice));
	checkCudaErrors(
			hipMemcpy(result->d_greyImage__, result->h_greyImage__,
					sizeof(unsigned char) * numPixels, hipMemcpyHostToDevice));
	result->CreateGreyscale();
	return result;
}

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
	  int blockId = gridDim.x * blockIdx.y + blockIdx.x;
	  int i = blockId * blockDim.x * blockDim.y + blockDim.x * threadIdx.y + threadIdx.x;

	  if(i < numRows * numCols)
	  {
	      greyImage[i] = .299f * rgbaImage[i].x + .587f * rgbaImage[i].y + .114f * rgbaImage[i].z;
	  }
}


void Image::CreateGreyscale()
{

	  const dim3 blockSize(32, 32, 1);  //TODO
	  const dim3 gridSize( 32, 32, 1);  //TODO
	  rgba_to_greyscale<<<gridSize, blockSize>>>(this->d_rgbaImage__, this->d_greyImage__, this->GetRowsCount(), this->GetColumnsCount());

	  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
	  checkCudaErrors(
				hipMemcpy(this->h_greyImage__, this->d_greyImage__,
						sizeof(unsigned char) * this->GetRowsCount() * this->GetColumnsCount(), hipMemcpyDeviceToHost));
}



void Image::SaveGrayscale(const std::string &filename)
{
	cv::Mat output(GetRowsCount(), GetColumnsCount(), CV_8UC1, (void*) h_greyImage__);

	//output the image
	cv::imwrite(filename.c_str(), output);
}

void Image::SaveRGBA(const std::string &filename)
{
	cv::Mat output(GetRowsCount(), GetColumnsCount(), CV_8UC4, (void*) h_rgbaImage__);
	bool cont = output.isContinuous();
	bool empt = output.empty();
	int channels = output.channels();
	cv::Mat bgr;
	cv::cvtColor(output, output, CV_RGB2BGR);
	//output the image
	cv::imwrite(filename.c_str(), output);
}




size_t Image::GetRowsCount() const
{
	return imageRGBA.rows;
}

size_t Image::GetColumnsCount() const
{
	return imageRGBA.cols;
}

uchar4 * Image::HostRGBA() const
{
	return h_rgbaImage__;
}
uchar4 * Image::DeviceRGBA() const
{
	return d_rgbaImage__;
}

unsigned char * Image::HostGreyscale() const
{
	return h_greyImage__;
}
unsigned char * Image::DeviceGreyscale() const
{
	return d_greyImage__;
}
