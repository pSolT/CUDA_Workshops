#include <iostream>
#include "utils.h"
#include <string>
#include <stdio.h>
#include "Image.cuh"
#include "SimpleFilter.cuh"
#include "ThresholdFilter.cuh"
#include "SobelFilter.cuh"
int main()
{

	std::string input_file;
	std::string output_RGBA_file;
	std::string output_Greyscale_file;
	Image * inputImage = Image::Load(input_file);

	//Skonfiguruj parametry wywołania
	int tileWidth;
	int tileHeight;
	int radius;;
	int blockWidth;
	int blockHeight;

	//Skonfiguruj wywołanie
	dim3 blockSize;
	dim3 gridSize;


	//Zastosuj filtr(y) na obrazek

	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	//Zapisz obrazek

	std::cout << "DONE!" << std::endl;

}
