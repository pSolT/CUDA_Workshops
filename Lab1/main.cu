#include "hip/hip_runtime.h"
#include "common/book.h"
#include "common/cpu_anim.h"

#define DIM 1024
#define PI 3.1415926535897932f

struct DataBlock{
	unsigned char *dev_bitmap;
	CPUAnimBitmap *bitmap;
};

// clean memory allocated on gpu
void cleanup(DataBlock *d){
	hipFree(d->dev_bitmap);
}

__global__ void kernel(unsigned char *ptr, int ticks)
{
	// Zmapuj współrzędne wątku w siatce na położenie konkretnego piksela na obrazku
	// int x =
	// int y =
	// int offset = (położenie względem pozątku tablicy ptr)
	

	// oblicz odległość od środka
	// float fx =
	// float fy =
	// float d = (odległość od środka)

	// oblicz kolor
	unsigned char grey = (unsigned char) (128.0f + 127.0f * cos(d/10.0f - ticks / 7.0f) / (d / 10.0f + 1.0f));
	//przypisz kolor pikselowi - 4 kanały !
}

void generate_frame(DataBlock *d, int ticks){
	//określ odpowiednie wymiary bloku
	//dim3 gridDim(?)
	//dim3 blockDim(?)

	//wywołaj kernel

	//skopiuj bitmapę z device na host

}

int main(void){
	DataBlock data;
	CPUAnimBitmap bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;
	
	HANDLE_ERROR(hipMalloc((void**)&data.dev_bitmap, bitmap.image_size()));
	
	bitmap.anim_and_exit((void (*)(void*,int))generate_frame, (void (*)(void*))cleanup);

}
