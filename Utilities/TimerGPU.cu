
#include "TimerGPU.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

struct CUDAEventTimer
{
    hipEvent_t     start;
    hipEvent_t     stop;
};

TimerGPU::TimerGPU()
{
	_timer = new CUDAEventTimer();
}


TimerGPU::~TimerGPU() { }

void TimerGPU::StartCounter()
{
    hipEventCreate(&((*_timer).start));
    hipEventCreate(&((*_timer).stop));
    hipEventRecord((*_timer).start,0);
}


float TimerGPU::GetCounter()
{
    float time;
    hipEventRecord((*_timer).stop, 0);
    hipEventSynchronize((*_timer).stop);
    hipEventElapsedTime(&time,(*_timer).start,(*_timer).stop);
    return time;
}
